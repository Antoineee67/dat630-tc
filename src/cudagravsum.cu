#include "hip/hip_runtime.h"
#include "treedefs.h"
#include <hip/hip_runtime.h>
#include <treecode.h>
#include <vector>
#include <vectdefs.h>
#include <vectmath.h>


#undef Update
#undef global

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/copy.h>
#include "hip/hip_vector_types.h"

typedef struct {
    vector acc;
    real phi;
} body_result;



#define EXPAND(x) x
#define CONCAT(a, b) a##b
#define MAKE_TYPE(base, dim) CONCAT(base, dim)

#ifndef DOUBLEPREC
#define cuda_vector MAKE_TYPE(float, NDIM)
#else
#define cuda_vector MAKE_TYPE(double, NDIM)
#endif


#if NDIM == 2
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1]})
#define CLR_CUDA_VECTOR(cv) ((cv) = {0, 0})
#elif NDIM == 3
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y, (cv).z})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1], (v)[2]})
#define CLR_CUDA_VECTOR(cv) ((cv) = {0, 0, 0})
#elif NDIM == 4
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y, (cv).z, (cv).w})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1], (v)[2], (v)[3]})
#define CLR_CUDA_VECTOR(cv) ((cv) = {0, 0, 0, 0})
#else
#error "NDIM must be 2, 3, or 4"
#endif

/*struct cuda_grav_pack
{
    uint32_t *body_cell_index_array; //ptr to body & cell array with indices that refer to correct body/cell.
    uint32_t *device_body_cell_index_array; //equivelent pointer but for the cuda device.
    uint32_t total_count; //length of above arrays.
    bodyptr current_body;

};*/

struct cuda_grav_pack_soa
{
    std::vector<uint32_t*> body_cell_index_array; //ptr to body & cell array with indices that refer to correct body/cell.
    std::vector<uint32_t*> device_body_cell_index_array; //equivelent pointer but for the cuda device.
    std::vector<size_t> device_body_cell_index_array_size; //byte size of above arrays
    uint32_t** device_body_cell_index_array_pointer_list; //Array to hold the device body cell index arrays.
    std::vector<uint32_t> total_count; //length of above arrays.
    uint32_t* device_total_count; //pointer to device of above array;
    std::vector<bodyptr> current_body;
    real *device_phi_out_list; //Our out arrays.
    vector *device_acc_out_list;
};

static hipStream_t localCudaStream;

static real *device_body_cell_mass_list;

static cuda_vector *device_body_cell_pos_list;

// static  cuda_grav_pack_soa cuda_grav_pack_list;


static thrust::host_vector<uint32_t> h_interact_vecs;

static thrust::host_vector<uint32_t> h_offset;

static thrust::host_vector<size_t> h_bodies_to_process;


// void new_cuda_grav_pack(cuda_grav_pack_soa *self, uint32_t *_body_cell_index_array, uint32_t *_device_body_cell_index_array, size_t _device_body_cell_index_array_size, uint32_t _total_count, bodyptr _current_body)
// {
//     self->body_cell_index_array.push_back(_body_cell_index_array);
//     self->device_body_cell_index_array.push_back(_device_body_cell_index_array);
//     self->device_body_cell_index_array_size.push_back(_device_body_cell_index_array_size);
//     self->total_count.push_back(_total_count);
//     self->current_body.push_back(_current_body);
// }





__global__ void cuda_node_calc_kernel(real eps2, uint32_t* interact_lists, uint32_t* offset, size_t* bodies_to_process,
    cuda_vector* pos_list, real* mass_list, real *out_phi_list, cuda_vector* out_acc_list, size_t n_bodies){

    uint32_t list_index = blockIdx.x*blockDim.x + threadIdx.x; //What body the gpu should work on corresponds to one element in the struct of arrays.

    if (list_index >= n_bodies)
        return;
    uint32_t end = offset[list_index];
    uint32_t start;
    if (list_index == 0){
        start = 0;
    }
    else{
        start = offset[list_index-1]+1;
    }

    real dr2, drab, phi_p, mr3i;
    //vector dr;
    cuda_vector dr;
    real local_phi0 = 0;
    cuda_vector local_acc0;
    CLR_CUDA_VECTOR(local_acc0);
    //CLRV(local_acc0);

    size_t current_body_index = bodies_to_process[list_index];

    for (uint32_t i = start; i < end; i++)
    {   
        size_t loop_body_index = interact_lists[i];
        // DOTPSUBV(dr2, dr, pos_list[loop_body_index], pos_list[current_body_index]);
        dr = pos_list[loop_body_index] - pos_list[current_body_index];
        dr2 = dot(dr, dr);
        dr2 += eps2;
        drab = rsqrt(dr2);
        phi_p = mass_list[loop_body_index]/drab;
        local_phi0 -= phi_p;
        mr3i = phi_p/dr2;
        //ADDMULVS(local_acc0, dr, mr3i);
        local_acc0 += dr * mr3i;
    }

    out_phi_list[current_body_index] = local_phi0;
    out_acc_list[current_body_index] = local_acc0;
}

void cuda_gravsum(bodyptr current_body, cell_ll_entry_t *cell_list_tail, cell_ll_entry_t *body_list_tail) {

    uint32_t cell_count = 0;
    cell_ll_entry_t *curr_list_entry = cell_list_tail;
    while (curr_list_entry->priv != nullptr) {
        h_interact_vecs.push_back(curr_list_entry->index + nbody);
        cell_count++;
        curr_list_entry = curr_list_entry->priv;
    }

    uint32_t body_count = 0;
    curr_list_entry = body_list_tail;
    while (curr_list_entry->priv != nullptr) {
        h_interact_vecs.push_back(curr_list_entry->index);
        body_count++;
        curr_list_entry = curr_list_entry->priv;
    }

    h_offset.push_back(h_interact_vecs.size()-1); // = end
    h_bodies_to_process.push_back((size_t)(current_body-bodytab));

}

//Will send cuda_grav_pack_list to gpu and calculate.
void cuda_gravsum_dispatch()
{
    size_t nBodiesToProcess = h_bodies_to_process.size();

    thrust::device_vector<uint32_t> d_interact_vecs;
    thrust::copy(thrust::cuda::par.on(localCudaStream), h_interact_vecs.begin(), h_interact_vecs.end(), d_interact_vecs.begin());
    thrust::device_vector<uint32_t> d_offset; 
    thrust::copy(thrust::cuda::par.on(localCudaStream), h_offset.begin(), h_offset.end(), d_offset.begin());
    thrust::device_vector<size_t> d_bodies_to_process; 
    thrust::copy(thrust::cuda::par.on(localCudaStream), h_bodies_to_process.begin(), h_bodies_to_process.end(), d_bodies_to_process.begin());
    thrust::device_vector<real> d_out_phi(nBodiesToProcess, 0);
    thrust::device_vector<cuda_vector> d_out_acc(nBodiesToProcess);

    uint32_t* d_interact_vecs_raw = thrust::raw_pointer_cast(d_interact_vecs.data());
    uint32_t* d_offset_raw = thrust::raw_pointer_cast(d_offset.data());
    size_t* d_bodies_to_process_raw = thrust::raw_pointer_cast(d_bodies_to_process.data());
    real* d_out_phi_raw = thrust::raw_pointer_cast(d_out_phi.data());
    cuda_vector* d_out_acc_raw = thrust::raw_pointer_cast(d_out_acc.data());


    //Start kernel
    int blocksize = 256;
    int nrGrids = (nBodiesToProcess + blocksize - 1)/blocksize;


    cuda_node_calc_kernel<<<nrGrids, blocksize, 0, localCudaStream>>>(eps2, 
        d_interact_vecs_raw, // 1D vector which links all interact lists
        d_offset_raw, // offset
        d_bodies_to_process_raw, 
        device_body_cell_pos_list, // pos
        device_body_cell_mass_list, // mass
        d_out_phi_raw, // phi
        d_out_acc_raw, // acc
        nBodiesToProcess);

    thrust::host_vector<real> h_out_phi;
    thrust::copy(thrust::cuda::par.on(localCudaStream), d_out_phi.begin(), d_out_phi.end(), h_out_phi.begin());
    thrust::host_vector<cuda_vector> h_out_acc;
    thrust::copy(thrust::cuda::par.on(localCudaStream), d_out_acc.begin(), d_out_acc.end(), h_out_acc.begin());

    

    //Apply phi0 and acc on bodies
    for (int i = 0; i < nBodiesToProcess; i++)
    {
        bodyptr current_bptr = bodytab+h_bodies_to_process[i];
        Phi(current_bptr) = h_out_phi[i];
        SETV(Acc(current_bptr), CUDA_VECTOR_TO_VECTOR(h_out_acc[i]));
        current_bptr->updated = TRUE;
    }
    //Free cuda memory
    // hipFreeAsync(cuda_grav_pack_list.device_phi_out_list, localCudaStream);
    // hipFreeAsync(cuda_grav_pack_list.device_acc_out_list, localCudaStream);
    // hipFreeAsync(cuda_grav_pack_list.device_body_cell_index_array_pointer_list, localCudaStream);
    // hipFreeAsync(cuda_grav_pack_list.device_total_count, localCudaStream);

    hipStreamSynchronize(localCudaStream);


}



void cuda_gravsum_init() {
    //TODO: Guard against/allow multiple calls?
    hipStreamCreate(&localCudaStream);
    printf("!!!!!!!!!! Only run on singlethread, struct of array is not thread safe\n");


    hipMalloc(&device_body_cell_mass_list, nbody * 2 * sizeof(real));
    hipMalloc(&device_body_cell_pos_list, nbody * 2 * sizeof(cuda_vector));



    real body_mass_list[nbody];

    for (int i = 0; i < nbody; i++) {
        body_mass_list[i] = Mass(&bodytab[i]);
    }

    hipMemcpy(device_body_cell_mass_list, body_mass_list, nbody * sizeof(real), hipMemcpyHostToDevice);
}

void cuda_update_body_cell_data() {
    vector body_cell_pos_list[nbody + ncell];

    for (int i = 0; i < nbody; i++) {
        SETV(body_cell_pos_list[i], Pos(&bodytab[i]));
    }

    for (int i = 0; i < ncell; i++) {
         SETV(body_cell_pos_list[i + nbody], Pos(&celltab[i]));
    }

    hipMemcpy(device_body_cell_pos_list, body_cell_pos_list, nbody * sizeof(cuda_vector), hipMemcpyHostToDevice);

    real cell_mass_list[ncell];

    for (int i = 0; i < ncell; i++) {
        cell_mass_list[i] = Mass(&celltab[i]);
    }
    hipMemcpy(device_body_cell_mass_list + nbody, cell_mass_list, ncell * sizeof(real), hipMemcpyHostToDevice);

    h_interact_vecs.clear();
    h_offset.clear();
    h_bodies_to_process.clear();
}
