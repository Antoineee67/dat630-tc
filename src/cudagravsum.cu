#include "hip/hip_runtime.h"
#include "treedefs.h"
#include <hip/hip_runtime.h>
#include <treecode.h>
#include <vector>
#include <vectdefs.h>
#include <vectmath.h>


#undef Update
#undef global

//#include "hip/hip_vector_types.h"
//#include <thrust/reduce.h>
//#include<thrust/execution_policy.h>
//#include <thrust/device_ptr.h>

typedef struct {
    vector acc;
    real phi;
} body_result;



#define EXPAND(x) x
#define CONCAT(a, b) a##b
#define MAKE_TYPE(base, dim) CONCAT(base, dim)

#ifndef DOUBLEPREC
#define cuda_vector MAKE_TYPE(float, NDIM)
#else
#define cuda_vector MAKE_TYPE(double, NDIM)
#endif


#if NDIM == 2
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1]})
#elif NDIM == 3
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y, (cv).z})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1], (v)[2]})
#elif NDIM == 4
#define CUDA_VECTOR_TO_VECTOR(cv) ((vector){(cv).x, (cv).y, (cv).z, (cv).w})
#define VECTOR_TO_CUDA_VECTOR(v) ((cuda_vector){(v)[0], (v)[1], (v)[2], (v)[3]})
#else
#error "NDIM must be 2, 3, or 4"
#endif

/*struct cuda_grav_pack
{
    uint32_t *body_cell_index_array; //ptr to body & cell array with indices that refer to correct body/cell.
    uint32_t *device_body_cell_index_array; //equivelent pointer but for the cuda device.
    uint32_t total_count; //length of above arrays.
    bodyptr current_body;

};*/

struct cuda_grav_pack_soa
{
    std::vector<uint32_t*> body_cell_index_array; //ptr to body & cell array with indices that refer to correct body/cell.
    std::vector<uint32_t*> device_body_cell_index_array; //equivelent pointer but for the cuda device.
    std::vector<size_t> device_body_cell_index_array_size; //byte size of above arrays
    uint32_t** device_body_cell_index_array_pointer_list; //Array to hold the device body cell index arrays.
    std::vector<uint32_t> total_count; //length of above arrays.
    uint32_t* device_total_count; //pointer to device of above array;
    std::vector<bodyptr> current_body;
    real *device_phi_out_list; //Our out arrays.
    vector *device_acc_out_list;
};

static hipStream_t localCudaStream;

static real *device_body_cell_mass_list;

static vector *device_body_cell_pos_list;

static  cuda_grav_pack_soa cuda_grav_pack_list;

void new_cuda_grav_pack(cuda_grav_pack_soa *self, uint32_t *_body_cell_index_array, uint32_t *_device_body_cell_index_array, size_t _device_body_cell_index_array_size, uint32_t _total_count, bodyptr _current_body)
{
    self->body_cell_index_array.push_back(_body_cell_index_array);
    self->device_body_cell_index_array.push_back(_device_body_cell_index_array);
    self->device_body_cell_index_array_size.push_back(_device_body_cell_index_array_size);
    self->total_count.push_back(_total_count);
    self->current_body.push_back(_current_body);
}





__global__ void cuda_node_calc_kernel(real eps2, uint32_t** body_cell_index_array_pointer, uint32_t* total_count_array,
    vector* body_cell_pos_list, real* body_cell_mass_list, real *phi_out_list, vector *acc_out_list, size_t maxSize){
    uint32_t current_body = blockIdx.x*blockDim.x + threadIdx.x; //What body the gpu should work on corresponds to one element in the struct of arrays.

    if (current_body > maxSize)
        return;
    uint32_t total_count = total_count_array[current_body];
    uint32_t* body_cell_index_array = body_cell_index_array_pointer[current_body];

    real dr2, drab, phi_p, mr3i;
    vector dr;
    real local_phi0 = 0;
    vector local_acc0;
    CLRV(local_acc0);

    for (int i = 0; i < total_count; i++)
    {
        uint32_t loop_body = body_cell_index_array[i];
        DOTPSUBV(dr2, dr, body_cell_pos_list[loop_body], body_cell_pos_list[current_body]);
        dr2 += eps2;
        drab = rsqrt(dr2);
        phi_p = body_cell_mass_list[loop_body]/drab;
        local_phi0 -= phi_p;
        mr3i = phi_p/dr2;
        ADDMULVS(local_acc0, dr, mr3i);
    }

    phi_out_list[current_body] = local_phi0;
    SETV(acc_out_list[current_body], local_acc0);
}

void cuda_gravsum(bodyptr current_body, cell_ll_entry_t *cell_list_tail, cell_ll_entry_t *body_list_tail) {

    uint32_t cell_count = 0;
    cell_ll_entry_t *curr_list_entry = cell_list_tail;
    while (curr_list_entry->priv != nullptr) {
        cell_count++;
        curr_list_entry = curr_list_entry->priv;
    }

    uint32_t body_count = 0;
    curr_list_entry = body_list_tail;
    while (curr_list_entry->priv != nullptr) {
        body_count++;
        curr_list_entry = curr_list_entry->priv;
    }

    uint32_t *body_cell_index_array = (uint32_t*) malloc((body_count + cell_count )* sizeof(uint32_t));
    curr_list_entry = body_list_tail;
    for (int i = 0; i < body_count; i++) {
        body_cell_index_array[i] = curr_list_entry->index;
        curr_list_entry = curr_list_entry->priv;
    }
    curr_list_entry = cell_list_tail;
    for (int i = 0; i < cell_count; i++) {
        body_cell_index_array[i + body_count] = curr_list_entry->index;
        curr_list_entry = curr_list_entry->priv;
    }

    const uint32_t total_count = body_count + cell_count;
    uint32_t *device_body_cell_index_list;
    hipMalloc(&device_body_cell_index_list,  total_count * sizeof(uint32_t));


    new_cuda_grav_pack(&cuda_grav_pack_list, body_cell_index_array, device_body_cell_index_list, total_count * sizeof(uint32_t),  total_count,current_body);


}

//Will send cuda_grav_pack_list to gpu and calculate.
void cuda_gravsum_dispatch()
{
    size_t bodiesToProcess = cuda_grav_pack_list.total_count.size();

    size_t failIndx;

    hipMalloc(&cuda_grav_pack_list.device_total_count, sizeof(uint32_t)*bodiesToProcess);
    hipMemcpyAsync(cuda_grav_pack_list.device_total_count, cuda_grav_pack_list.total_count.data(),
        bodiesToProcess, hipMemcpyHostToDevice, localCudaStream);

    hipMalloc(&cuda_grav_pack_list.device_body_cell_index_array_pointer_list, sizeof(uint32_t*)*cuda_grav_pack_list.device_body_cell_index_array.size());
    hipMemcpyAsync(cuda_grav_pack_list.device_body_cell_index_array_pointer_list, cuda_grav_pack_list.device_body_cell_index_array.data(),
        bodiesToProcess, hipMemcpyHostToDevice, localCudaStream);


    hipMemcpyAttributes attrList[] = {hipMemcpyAttributes{hipMemcpySrcAccessOrderAny, {}, {}, hipMemcpyHostToDevice}};
    size_t attrIndx[] = {0};

    hipMemcpyBatchAsync(cuda_grav_pack_list.device_body_cell_index_array.data(), cuda_grav_pack_list.body_cell_index_array.data(),
         cuda_grav_pack_list.device_body_cell_index_array_size.data(), bodiesToProcess, attrList,
         attrIndx, 1, &failIndx, localCudaStream
         );
    printf("%lu", failIndx);


    hipMalloc(&cuda_grav_pack_list.device_phi_out_list, bodiesToProcess * sizeof(real));
    hipMalloc(&cuda_grav_pack_list.device_acc_out_list, bodiesToProcess * sizeof(vector));

    //Start kernel
    int blocksize = 256;
    int nrGrids = (bodiesToProcess + blocksize - 1)/blocksize;




    cuda_node_calc_kernel<<<nrGrids, blocksize, 0, localCudaStream>>>(eps2, cuda_grav_pack_list.device_body_cell_index_array_pointer_list,
       cuda_grav_pack_list.device_total_count,
       device_body_cell_pos_list,
       device_body_cell_mass_list,
       cuda_grav_pack_list.device_phi_out_list,
       cuda_grav_pack_list.device_acc_out_list,
       bodiesToProcess);

    real *phi_out_list = (real*) malloc(bodiesToProcess* sizeof(real));
    vector *acc_out_list = (vector*) malloc(bodiesToProcess*sizeof(vector));

    hipMemcpyAsync(phi_out_list, cuda_grav_pack_list.device_phi_out_list, bodiesToProcess*sizeof(real), hipMemcpyDeviceToHost, localCudaStream);
    hipMemcpyAsync(acc_out_list, cuda_grav_pack_list.device_acc_out_list, bodiesToProcess*sizeof(vector), hipMemcpyDeviceToHost, localCudaStream);

    //Apply phi0 and acc on bodies
    for (int i = 0; i < bodiesToProcess; i++)
    {
        Phi(cuda_grav_pack_list.current_body[i]) = phi_out_list[i];
        SETV(Acc(cuda_grav_pack_list.current_body[i]), acc_out_list[i]);
        cuda_grav_pack_list.current_body[i]->updated = TRUE;
        hipFreeAsync(cuda_grav_pack_list.device_acc_out_list[i], localCudaStream);

    }
    //Free cuda memory
    hipFreeAsync(cuda_grav_pack_list.device_phi_out_list, localCudaStream);
    hipFreeAsync(cuda_grav_pack_list.device_acc_out_list, localCudaStream);
    hipFreeAsync(cuda_grav_pack_list.device_body_cell_index_array_pointer_list, localCudaStream);
    hipFreeAsync(cuda_grav_pack_list.device_total_count, localCudaStream);



    hipStreamSynchronize(localCudaStream);


}



void cuda_gravsum_init() {
    //TODO: Guard against/allow multiple calls?
    hipStreamCreate(&localCudaStream);
    printf("!!!!!!!!!! Only run on singlethread, struct of array is not thread safe");


    hipMalloc(&device_body_cell_mass_list, nbody * 2 * sizeof(real));
    hipMalloc(&device_body_cell_pos_list, nbody * 2 * sizeof(vector));



    real body_mass_list[nbody];

    for (int i = 0; i < nbody; i++) {
        body_mass_list[i] = Mass(&bodytab[i]);
    }

    hipMemcpy(device_body_cell_mass_list, body_mass_list, nbody * sizeof(real), hipMemcpyHostToDevice);
}

void cuda_update_body_cell_data() {
    vector body_cell_pos_list[nbody + ncell];

    for (int i = 0; i < nbody; i++) {
        SETV(body_cell_pos_list[i], Pos(&bodytab[i]));
    }

    for (int i = 0; i < ncell; i++) {
         SETV(body_cell_pos_list[i + nbody], Pos(&celltab[i]));
    }

    hipMemcpy(device_body_cell_pos_list, body_cell_pos_list, nbody * sizeof(vector), hipMemcpyHostToDevice);

    real cell_mass_list[ncell];

    for (int i = 0; i < ncell; i++) {
        cell_mass_list[i] = Mass(&celltab[i]);
    }
    hipMemcpy(device_body_cell_mass_list + nbody, cell_mass_list, ncell * sizeof(real), hipMemcpyHostToDevice);
}
